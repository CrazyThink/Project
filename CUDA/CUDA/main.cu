#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <string>
#include <array>
#include <vector>
#include <iterator>
#include <Windows.h>
#include <assert.h>
#include <omp.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "DS_timer.h"
#include "hip/hip_runtime.h"
#include ""

using namespace std;

#define THREAD 8
#define THREAD_IN_BLOCK 256

uint32_t fileSize;
uint32_t dataOffset;
uint32_t width;
uint32_t height;
uint32_t depth;
int dataSize;

__global__ void grayFilter(char* img, int dataSize) {
	int tid = (blockIdx.x * blockDim.x + threadIdx.x) * 3;
	if (tid > dataSize) return;

	double R = img[tid + 2] & 0xff;
	double G = img[tid + 1] & 0xff;
	double B = img[tid    ] & 0xff;

	char temp = R * .299f + G * .587f + B * .114f;
	img[tid + 2] = temp;	//R
	img[tid + 1] = temp;	//G
	img[tid    ] = temp;	//B
}

__global__ void brightFilter(char* img, int dataSize) {
	int tid = (blockIdx.x * blockDim.x + threadIdx.x) * 3;
	if (tid > dataSize) return;

	double R = img[tid + 2] & 0xff;
	double G = img[tid + 1] & 0xff;
	double B = img[tid    ] & 0xff;

	img[tid + 2] = (R + R * .2f) > 255 ? 255 : (R + R * .2f);
	img[tid + 1] = (G + G * .2f) > 255 ? 255 : (G + G * .2f);
	img[tid    ] = (B + B * .2f) > 255 ? 255 : (B + B * .2f);
}

__global__ void sepiaFilter(char* img, int dataSize) {
	int tid = (blockIdx.x * blockDim.x + threadIdx.x) * 3;
	if (tid > dataSize) return;

	double R = img[tid + 2] & 0xff;
	double G = img[tid + 1] & 0xff;
	double B = img[tid] & 0xff;

	double R_temp = R * .393f + G * .769f + B * .189f;
	double G_temp = R * .349f + G * .686f + B * .168f;
	double B_temp = R * .272f + G * .534f + B * .131f;

	img[tid + 2] = R_temp > 255 ? 255 : R_temp;
	img[tid + 1] = G_temp > 255 ? 255 : G_temp;
	img[tid] = B_temp > 255 ? 255 : B_temp;
}
__global__ void customFilter(char* img, int dataSize) {
	int tid = (blockIdx.x * blockDim.x + threadIdx.x) * 3;
	if (tid > dataSize) return;

	double R = img[tid + 2] & 0xff;
	double G = img[tid + 1] & 0xff;
	double B = img[tid] & 0xff;

	double R_temp = R;
	double G_temp = G;
	double B_temp = B;

	const double rate = .2f;
	int standard = 15;		//���� ��� ��

	if (R > 160 && G > 160 && B > 160) {
		standard = 20;
	}
	if (R > B && G > B) {	//B�� ���� ���� ��
		R_temp = R + R * rate;
		G_temp = G + G * rate;
	}
	if (R > G && B > G) {	//G�� ���� ���� ��
		R_temp = R + R * rate;
		B_temp = B + B * rate;
	}
	if (G > R && B > R) {	//R�� ���� ���� ��
		G_temp = G + G * rate;
		B_temp = B + B * rate;
	}
	if (R > B && R > G) {	//R�� ���� Ŭ ��
		R_temp = R + R * rate;
		G_temp = G + G * rate;
		B_temp = B;
		if (G_temp < B_temp) B_temp = B + B * rate;
	}
	if (G > R && G > B) {	//G�� ���� Ŭ ��
		R_temp = R + R * rate;
		G_temp = G + G * rate;
		B_temp = B;
		if (R_temp < B_temp) B_temp = B + B * rate;
	}
	if (abs(R - G) < standard && abs(G - B) < standard && abs(R - B) < standard) {
		R_temp = R + R * rate;
		G_temp = G + G * rate;
		B_temp = B;
	}
	else if (B > R && B > G && abs(G - B) < standard && abs(R - B) < standard * 2) {
		R_temp = R + R * rate;
		G_temp = G + G * rate;
		B_temp = B;
	}
	else if (B > R && B > G) {//B�� ���� Ŭ ��
		R_temp = R;
		G_temp = G;
		B_temp = B + B * rate;
		if (R_temp > G_temp) R_temp = R + R * rate;
		else G_temp = G + G * rate;
	}

	img[tid + 2] = R_temp > 255 ? 255 : R_temp;
	img[tid + 1] = G_temp > 255 ? 255 : G_temp;
	img[tid] = B_temp > 255 ? 255 : B_temp;
}

inline vector<char> grayFilter(vector<char> img) {
	#pragma omp parallel for num_threads(THREAD)
	for (int i = dataSize - 3; i >= 0; i -= 3){
		double R = img[i + 2] & 0xff;
		double G = img[i + 1] & 0xff;
		double B = img[i] & 0xff;

		char temp = R * .299f + G * .587f + B * .114f;
		img[i + 2] = temp;	//R
		img[i + 1] = temp;	//G
		img[i] = temp;	//B
	}
	return img;
}

inline vector<char> brightFilter(vector<char> img) {
	#pragma omp parallel for num_threads(THREAD)
	for (int i = dataSize - 3; i >= 0; i -= 3) {
		double R = img[i + 2] & 0xff;
		double G = img[i + 1] & 0xff;
		double B = img[i] & 0xff;

		img[i + 2] = (R + R * .2f) > 255 ? 255 : (R + R * .2f);
		img[i + 1] = (G + G * .2f) > 255 ? 255 : (G + G * .2f);
		img[i] = (B + B * .2f) > 255 ? 255 : (B + B * .2f);
	}
	return img;
}

inline vector<char> sepiaFilter(vector<char> img) {
	#pragma omp parallel for num_threads(THREAD)
	for (int i = dataSize - 3; i >= 0; i -= 3){
		double R = img[i + 2] & 0xff;
		double G = img[i + 1] & 0xff;
		double B = img[i] & 0xff;

		double R_temp = R * .393f + G * .769f + B * .189f;
		double G_temp = R * .349f + G * .686f + B * .168f;
		double B_temp = R * .272f + G * .534f + B * .131f;

		img[i + 2] = R_temp > 255 ? 255 : R_temp;
		img[i + 1] = G_temp > 255 ? 255 : G_temp;
		img[i] = B_temp > 255 ? 255 : B_temp;
	}
	return img;
}

inline vector<char> customFilter(vector<char> img) {
	#pragma omp parallel for num_threads(THREAD)
	for (int i = dataSize - 3; i >= 0; i -= 3){
		double R = img[i + 2] & 0xff;
		double G = img[i + 1] & 0xff;
		double B = img[i] & 0xff;

		double R_temp = R;
		double G_temp = G;
		double B_temp = B;

		const double rate = .2f;
		int standard = 15;		//���� ��� ��

		if (R > 160 && G > 160 && B > 160) {
			standard = 20;
		}

		if (R > B && G > B) {	//B�� ���� ���� ��
			R_temp = R + R * rate;
			G_temp = G + G * rate;
		}
		if (R > G && B > G) {	//G�� ���� ���� ��
			R_temp = R + R * rate;
			B_temp = B + B * rate;
		}
		if (G > R && B > R) {	//R�� ���� ���� ��
			G_temp = G + G * rate;
			B_temp = B + B * rate;
		}
		if (R > B && R > G) {	//R�� ���� Ŭ ��
			R_temp = R + R * rate;
			G_temp = G + G * rate;
			B_temp = B;
			if (G_temp < B_temp) B_temp = B + B * rate;
		}
		if (G > R && G > B) {	//G�� ���� Ŭ ��
			R_temp = R + R * rate;
			G_temp = G + G * rate;
			B_temp = B;
			if (R_temp < B_temp) B_temp = B + B * rate;
		}
		if (abs(R - G) < standard && abs(G - B) < standard && abs(R - B) < standard) {
			R_temp = R + R * rate;
			G_temp = G + G * rate;
			B_temp = B;
		}
		else if (B > R && B > G && abs(G - B) < standard && abs(R - B) < standard * 2) {
			R_temp = R + R * rate;
			G_temp = G + G * rate;
			B_temp = B;
		}
		else if (B > R && B > G) {//B�� ���� Ŭ ��
			R_temp = R;
			G_temp = G;
			B_temp = B + B * rate;
			if (R_temp > G_temp) R_temp = R + R * rate;
			else G_temp = G + G * rate;
		}


		img[i + 2] = R_temp > 255 ? 255 : R_temp;
		img[i + 1] = G_temp > 255 ? 255 : G_temp;
		img[i] = B_temp > 255 ? 255 : B_temp;
	}
	return img;
}

vector<char> readBMP(const string &file)
{
	static constexpr size_t HEADER_SIZE = 54;

	ifstream bmp(file, ios::binary);

	array<char, HEADER_SIZE> header;
	bmp.read(header.data(), header.size());

	fileSize	= *reinterpret_cast<uint32_t *>(&header[ 2]);
	dataOffset	= *reinterpret_cast<uint32_t *>(&header[10]);
	width		= *reinterpret_cast<uint32_t *>(&header[18]);
	height		= *reinterpret_cast<uint32_t *>(&header[22]);
	depth		= *reinterpret_cast<uint16_t *>(&header[28]);

	cout << "fileSize: " << fileSize << endl;
	cout << "dataOffset: " << dataOffset << endl;
	cout << "width: " << width << endl;
	cout << "height: " << height << endl;
	cout << "depth: " << depth << "-bit" << endl;

	vector<char> img(dataOffset - HEADER_SIZE);
	bmp.read(img.data(), img.size());

	dataSize = ((width * 3 + 3) & (~3)) * height;
	img.resize(dataSize);
	bmp.read(img.data(), img.size());

	return img;
}

bool SaveImage(const string& szPathName, const vector<char>& lpBits, int w, int h) {
	ofstream pFile(szPathName, ios_base::binary);
	if (!pFile.is_open()) return false;

	BITMAPINFOHEADER bmih;
	bmih.biSize = sizeof(BITMAPINFOHEADER);
	bmih.biWidth = w;
	bmih.biHeight = h;
	bmih.biPlanes = 1;
	bmih.biBitCount = 24;
	bmih.biCompression = BI_RGB;
	bmih.biSizeImage = w * h * 3;

	BITMAPFILEHEADER bmfh;
	int nBitsOffset = sizeof(BITMAPFILEHEADER) + bmih.biSize;
	LONG lImageSize = bmih.biSizeImage;
	LONG lFileSize = nBitsOffset + lImageSize;
	bmfh.bfType = 'B' + ('M' << 8);
	bmfh.bfOffBits = nBitsOffset;
	bmfh.bfSize = lFileSize;
	bmfh.bfReserved1 = bmfh.bfReserved2 = 0;

	pFile.write((const char*)&bmfh, sizeof(BITMAPFILEHEADER));
	UINT nWrittenFileHeaderSize = pFile.tellp();

	pFile.write((const char*)&bmih, sizeof(BITMAPINFOHEADER));
	UINT nWrittenInfoHeaderSize = pFile.tellp();

	pFile.write(&lpBits[0], lpBits.size());
	UINT nWrittenDIBDataSize = pFile.tellp();
	pFile.close();

	return true;
}

int main() {
	//Ÿ�̸� ����
	DS_timer timer(2);
	timer.initTimers();

	//���� ����
	string filename;
	string completeName[4] = { "_grayFilter","_sepiaFilter","_brightFilter","_customFilter" };
	int num;

	//���� �б�
	printf("��ȯ�� ���� �̸� (Ȯ���� ����) : ");
	cin >> filename;
	vector<char> img = readBMP(filename + ".bmp");
	thrust::device_vector<char> d_img(img);
	dim3 dimGrid(((dataSize / 3 + 1) / THREAD_IN_BLOCK) + 1, 1, 1);
	dim3 dimBlock(THREAD_IN_BLOCK, 1, 1);

	//��ٷ� �� ����
	char* raw_ptr = thrust::raw_pointer_cast(d_img.data());

	printf("������ ��� �о����ϴ�. �Ʒ� �� �ϳ��� �����ϼ���.\n");
	printf("1 : grayFilter\n");
	printf("2 : sepiaFilter\n");
	printf("3 : brightFilter\n");
	printf("4 : customFilter\n");
	scanf("%d", &num);

	//OpenMP Version
	timer.onTimer(0);
	switch (num) {
		case 1:	img = grayFilter(img); break;
		case 2: img = sepiaFilter(img); break;
		case 3: img = brightFilter(img); break;
		case 4: img = customFilter(img); break;
		default: printf("�߸��� ���� �Է� �޾ҽ��ϴ�.\n"); return 0;
	}
	timer.offTimer(0);
	printf("CPU Complete!!\n");

	//CUDA Version
	timer.onTimer(1);
	switch (num) {
		case 1:	grayFilter <<< dimGrid, dimBlock >>>(raw_ptr, dataSize); break;
		case 2: sepiaFilter <<< dimGrid, dimBlock >>>(raw_ptr, dataSize); break;
		case 3: brightFilter << < dimGrid, dimBlock >> >(raw_ptr, dataSize); break;
		case 4: customFilter << < dimGrid, dimBlock >> >(raw_ptr, dataSize); break;
		default: printf("�߸��� ���� �Է� �޾ҽ��ϴ�.\n"); return 0;
	}
	hipDeviceSynchronize();
	timer.offTimer(1);
	printf("GPU Complete!!\n");

	//��ٿ��� �� ������
	thrust::copy(d_img.begin(), d_img.end(), img.begin());

	filename += completeName[num - 1];
	filename += ".bmp";
	cout << "����� ���� �̸� : " << filename << endl;
	SaveImage(filename, img, width, height);

	timer.printTimer();

	return 0;
}